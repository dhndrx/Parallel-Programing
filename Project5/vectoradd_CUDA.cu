/*
Vector addition code for CS 4380 / CS 5351

Copyright (c) 2019 Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <sys/time.h>

static const int ThreadsPerBlock = 512;

static __global__ void vadd(int c[], const int a[], const int b[], const int size)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  // perform vector addition
  if (i < size) {
    c[i] = a[i] + b[i];
  }
}

static void CheckCuda()
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d: %s\n", e, hipGetErrorString(e));
    exit(-1);
  }
}

int main(int argc, char *argv[])
{
  printf("Vector addition v1.0\n");

  // check command line
  if (argc != 2) {fprintf(stderr, "USAGE: %s vector_size\n", argv[0]); exit(-1);}
  const int size = atoi(argv[1]);
  if (size < 8) {fprintf(stderr, "ERROR: vector_size must be at least 8\n"); exit(-1);}
  printf("vector size: %ld\n", size);

  // allocate vectors
  int* const a = new int [size];
  int* const b = new int [size];
  int* const c = new int [size];

  // initialize vectors
  for (int i = 0; i < size; i++) a[i] = i;
  for (int i = 0; i < size; i++) b[i] = size - i;
  for (int i = 0; i < size; i++) c[i] = -1;

  // allocate vectors on GPU
  int* d_a;
  int* d_b;
  int* d_c;
  if (hipSuccess != hipMalloc((void **)&d_a, sizeof(int) * size)) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}
  if (hipSuccess != hipMalloc((void **)&d_b, sizeof(int) * size)) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}
  if (hipSuccess != hipMalloc((void **)&d_c, sizeof(int) * size)) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}

  // initialize vectors on GPU
  if (hipSuccess != hipMemcpy(d_a, a, sizeof(int) * size, hipMemcpyHostToDevice)) {fprintf(stderr, "ERROR: copying to device failed\n"); exit(-1);}
  if (hipSuccess != hipMemcpy(d_b, b, sizeof(int) * size, hipMemcpyHostToDevice)) {fprintf(stderr, "ERROR: copying to device failed\n"); exit(-1);}
  if (hipSuccess != hipMemcpy(d_c, c, sizeof(int) * size, hipMemcpyHostToDevice)) {fprintf(stderr, "ERROR: copying to device failed\n"); exit(-1);}

  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  // execute timed code
  vadd<<<(size + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(d_c, d_a, d_b, size);
  hipDeviceSynchronize();

  // end time
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0;
  printf("compute time: %.4f s\n", runtime);

  // get result from GPU
  CheckCuda();
  if (hipSuccess != hipMemcpy(c, d_c, sizeof(int) * size, hipMemcpyDeviceToHost)) {fprintf(stderr, "ERROR: copying from device failed\n"); exit(-1);}

  // verify result
  for (int i = 0; i < size; i++) {
    if (c[i] != size) {fprintf(stderr, "ERROR: incorrect result\n"); exit(-1);}
  }
  printf("verification passed\n");

  // clean up
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  delete [] a;
  delete [] b;
  delete [] c;
  return 0;
}
